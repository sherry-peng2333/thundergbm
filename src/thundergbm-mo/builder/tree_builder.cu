//
// Created by jiashuai on 19-1-23.
//

#include <builder/tree_builder.h>
#include "util/multi_device.h"
#include "util/device_lambda.cuh"

void TreeBuilder::update_tree() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto& sp = this->sp[device_id];
        auto& tree = this->trees[device_id];
        SyncArray<GHPair> &sp_fea_missing_gh = this->sp_fea_missing_gh[device_id];
        SyncArray<GHPair> &sp_rch_sum_gh = this->sp_rch_sum_gh[device_id];
        auto sp_data = sp.device_data();
        LOG(DEBUG) << sp;
        int n_nodes_in_level = sp.size();

        Tree::TreeNode *nodes_data = tree.nodes.device_data();
        float_type rt_eps = param.rt_eps;
        float_type lambda = param.lambda;
        if(multi_outputs){
            int d_outputs = this->d_outputs_;
            GHPair *sum_gh_pair_mo_data = tree.sum_gh_pair_mo.device_data();
            float_type *base_weight_mo_data = tree.base_weight_mo.device_data();
            auto sp_fea_missing_gh_data = sp_fea_missing_gh.device_data();
            auto sp_rch_sum_gh_data = sp_rch_sum_gh.device_data();
            // when it is multi_outputs
            device_loop(n_nodes_in_level, [=]__device__(int i) {
                float_type best_split_gain = sp_data[i].gain;
                if (best_split_gain > rt_eps) {
                    //do split
                    if (sp_data[i].nid == -1) return;
                    int nid = sp_data[i].nid;
                    Tree::TreeNode &node = nodes_data[nid];
                    node.gain = best_split_gain;
                    int lid = node.lch_index;
                    int rid = node.rch_index;
                    Tree::TreeNode &lch = nodes_data[lid];//left child
                    Tree::TreeNode &rch = nodes_data[rid];//right child
                    lch.is_valid = true;
                    rch.is_valid = true;
                    node.split_feature_id = sp_data[i].split_fea_id;
                    //todo process begin
                    node.split_value = sp_data[i].fval;
                    node.split_bid = sp_data[i].split_bid;
                    for(int j = 0; j < d_outputs; j++){
                        sum_gh_pair_mo_data[rid*d_outputs+j] = sp_rch_sum_gh_data[i*d_outputs+j];
                    }
                    if (sp_data[i].default_right) {
                        for(int j = 0; j < d_outputs; j++){
                            sum_gh_pair_mo_data[rid*d_outputs+j] = sum_gh_pair_mo_data[rid*d_outputs+j] + sp_fea_missing_gh_data[i*d_outputs+j];
                        }
                        node.default_right = true;
                    }
                    for(int j = 0; j < d_outputs; j++){
                        sum_gh_pair_mo_data[lid*d_outputs+j] = sum_gh_pair_mo_data[nid*d_outputs+j] - sum_gh_pair_mo_data[rid*d_outputs+j];
                    }
                    for(int j = 0; j < d_outputs; j++){
                        base_weight_mo_data[lid*d_outputs+j] = -sum_gh_pair_mo_data[lid*d_outputs+j].g / (sum_gh_pair_mo_data[lid*d_outputs+j].h + lambda);
                        base_weight_mo_data[rid*d_outputs+j] = -sum_gh_pair_mo_data[rid*d_outputs+j].g / (sum_gh_pair_mo_data[rid*d_outputs+j].h + lambda);
                    }
                } else {
                    //set leaf
                    if (sp_data[i].nid == -1) return;
                    int nid = sp_data[i].nid;
                    Tree::TreeNode &node = nodes_data[nid];
                    node.is_leaf = true;
                    nodes_data[node.lch_index].is_valid = false;
                    nodes_data[node.rch_index].is_valid = false;
                }
            });
        }
        else{
            device_loop(n_nodes_in_level, [=]__device__(int i) {
                float_type best_split_gain = sp_data[i].gain;
                if (best_split_gain > rt_eps) {
                    //do split
                    if (sp_data[i].nid == -1) return;
                    int nid = sp_data[i].nid;
                    Tree::TreeNode &node = nodes_data[nid];
                    node.gain = best_split_gain;

                    Tree::TreeNode &lch = nodes_data[node.lch_index];//left child
                    Tree::TreeNode &rch = nodes_data[node.rch_index];//right child
                    lch.is_valid = true;
                    rch.is_valid = true;
                    node.split_feature_id = sp_data[i].split_fea_id;
                    GHPair p_missing_gh = sp_data[i].fea_missing_gh;
                    //todo process begin
                    node.split_value = sp_data[i].fval;
                    node.split_bid = sp_data[i].split_bid;
                    rch.sum_gh_pair = sp_data[i].rch_sum_gh;
                    if (sp_data[i].default_right) {
                        rch.sum_gh_pair = rch.sum_gh_pair + p_missing_gh;
                        node.default_right = true;
                    }
                    lch.sum_gh_pair = node.sum_gh_pair - rch.sum_gh_pair;
                    lch.calc_weight(lambda);
                    rch.calc_weight(lambda);
                } else {
                    //set leaf
                    if (sp_data[i].nid == -1) return;
                    int nid = sp_data[i].nid;
                    Tree::TreeNode &node = nodes_data[nid];
                    node.is_leaf = true;
                    nodes_data[node.lch_index].is_valid = false;
                    nodes_data[node.rch_index].is_valid = false;
                }
            });
        }
        LOG(DEBUG) << tree.nodes;
    });
}

void TreeBuilder::predict_in_training(int k) {
    if(multi_outputs){
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            auto y_predict_data = y_predict[device_id].device_data() + k * n_instances;
            auto nid_data = ins2node_id[device_id].device_data();
            const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
            auto base_weight_data = trees[device_id].base_weight_mo.device_data();
            int d_outputs_ = this->d_outputs_;
            auto lr = param.learning_rate;
            device_loop(n_instances, [=]__device__(int i) {
                int nid = nid_data[i];
                while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
                for(int j = 0; j < d_outputs_; j++){
                    y_predict_data[i*d_outputs_+j] += lr * base_weight_data[nid*d_outputs_+j];
                }
            });
        });
    }
    else{
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            auto y_predict_data = y_predict[device_id].device_data() + k * n_instances;
            auto nid_data = ins2node_id[device_id].device_data();
            const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
            auto lr = param.learning_rate;
            device_loop(n_instances, [=]__device__(int i) {
                int nid = nid_data[i];
                while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
                y_predict_data[i] += lr * nodes_data[nid].base_weight;
            });
        });
    }

}

void TreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    int n_available_device;
    hipGetDeviceCount(&n_available_device);
    CHECK_GE(n_available_device, param.n_device) << "only " << n_available_device
                                                 << " GPUs available; please set correct number of GPUs to use";
    FunctionBuilder::init(dataset, param);
    this->n_instances = dataset.n_instances_;
    this->d_outputs_ = dataset.d_outputs_;
    this->multi_outputs = param.multi_outputs;
    trees = vector<Tree>(param.n_device);
    ins2node_id = MSyncArray<int>(param.n_device, n_instances);
    sp = MSyncArray<SplitPoint>(param.n_device);
    has_split = vector<bool>(param.n_device);
    int n_outputs = param.num_class * n_instances * param.d_outputs_;
    y_predict = MSyncArray<float_type>(param.n_device, n_outputs);
    gradients = MSyncArray<GHPair>(param.n_device, n_instances * param.d_outputs_);
    if(param.multi_outputs){
        sp_fea_missing_gh = MSyncArray<GHPair>(param.n_device);
        sp_rch_sum_gh = MSyncArray<GHPair>(param.n_device);
    }
}

void TreeBuilder::ins2node_id_all_reduce(int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = ins2node_id.front().device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(ins2node_id[d]);
            device_loop(n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        ins2node_id[device_id].copy_from(ins2node_id.front());
    });
}

void TreeBuilder::split_point_all_reduce(int depth) {
    TIMED_FUNC(timerObj);
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = sp.front().host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = sp[device_id].host_data();
        for (int j = 0; j < sp[device_id].size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos])
                global_sp_data[global_pos] = local_sp_data[j];
            else
                global_sp_data[global_pos] = (global_sp_data[global_pos].gain >= local_sp_data[j].gain)
                                             ?
                                             global_sp_data[global_pos] : local_sp_data[j];
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
       sp[device_id].copy_from(sp.front());
    });
    LOG(DEBUG) << "global best split point = " << sp.front();
}

vector<Tree> TreeBuilder::build_approximate(const MSyncArray<GHPair> &gradients) {
    vector<Tree> trees(param.tree_per_rounds);
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        this->shards[device_id].column_sampling(param.column_sampling_rate);
    });
    float_type build_hist_total_time = 0;
    float_type subtract_time = 0;
    for (int k = 0; k < param.tree_per_rounds; ++k) {
        Tree &tree = trees[k];
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->ins2node_id[device_id].resize(n_instances);
            this->gradients[device_id].set_device_data(const_cast<GHPair *>(gradients[device_id].device_data() + k * n_instances));
            this->trees[device_id].init2(this->gradients[device_id], param);
        });

        for (int level = 0; level < param.depth; ++level) {
            DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
                find_split_mo(level, device_id);
            });
            split_point_all_reduce(level);
            {
                TIMED_SCOPE(timerObj, "apply sp");
                update_tree();
                update_ins2node_id();
                {
                    LOG(TRACE) << "gathering ins2node id";
                    //get final result of the reset instance id to node id
                    bool has_split = false;
                    for (int d = 0; d < param.n_device; d++) {
                        has_split |= this->has_split[d];
                    }
                    if (!has_split) {
                        LOG(INFO) << "no splittable nodes, stop";
                        break;
                    }
                }
                ins2node_id_all_reduce(level);
            }
        }
//        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
//            this->trees[device_id].prune_self(param.gamma);
//        });
        predict_in_training(k);
        tree.nodes.resize(this->trees.front().nodes.size());
        tree.nodes.copy_from(this->trees.front().nodes);
        tree.sum_gh_pair_mo.resize(this->trees.front().sum_gh_pair_mo.size());
        tree.sum_gh_pair_mo.copy_from(this->trees.front().sum_gh_pair_mo);
        tree.base_weight_mo.resize(this->trees.front().base_weight_mo.size());
        tree.base_weight_mo.copy_from(this->trees.front().base_weight_mo);
        tree.d_outputs_ = this->trees.front().d_outputs_;
        build_hist_total_time += this->build_hist_time;
        subtract_time += this->subtract_time;
//        string s = tree.dump(param.depth);
//        LOG(INFO) << "TREE:" << s;
    }
    LOG(INFO) << "time for atom operation of building hist: " << build_hist_total_time;
    LOG(INFO) << "time for subtract operation: " << subtract_time;
    return trees;
}
