#include "hip/hip_runtime.h"
//
// Created by qinbin on 2018/5/9.
//

#include "hist_cut.h"
#include "quantile_sketch.h"
#include "syncarray.h"
#include <sstream>
#include <omp.h>
#include <hist_cut.h>
#include <util/cub_wrapper.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>
#include <climits>
#include "util/device_lambda.cuh"
#include "thrust/unique.h"

/**
 * not fast but need less memory
 */
void HistCut::get_cut_points2(SparseColumns &columns, int max_num_bins, int n_instances) {
    LOG(INFO) << "Getting cut points...";
    int n_column = columns.n_column;
    SyncArray<float> unique_vals(n_column * n_instances);
    SyncArray<int> tmp_row_ptr(n_column + 1);

    SyncArray<int> tmp_params(2);  // [0] --> num_cut_points, [1] --> max_num_bins
    int h_tmp_params[2] = {0, max_num_bins};
    tmp_params.copy_from(h_tmp_params, 2);

    // get the original csc data
    auto csc_val_data = columns.csc_val.device_data();
    auto csc_col_ptr_data = columns.csc_col_ptr.device_data();
    auto unique_vals_data = unique_vals.device_data();
    auto tmp_row_ptr_data = tmp_row_ptr.device_data();
    auto tmp_params_data = tmp_params.device_data();

    // start to get cut points
    device_loop(n_column, [=] __device__(int fid){
        int col_start = csc_col_ptr_data[fid];
        int col_len = csc_col_ptr_data[fid+1] - col_start;
        auto val_data = csc_val_data + col_start;
        auto unique_start = unique_vals_data + fid*n_instances;  // notice here
        int unique_len = thrust::unique_copy(thrust::device, val_data, val_data + col_len, unique_start) - unique_start;
        int n_cp = (unique_len <= tmp_params_data[1]) ? unique_len : tmp_params_data[1];
        tmp_row_ptr_data[fid+1] = unique_len;
        atomicAdd(&tmp_params_data[0], n_cp);
    });

    // merge the cut points
    tmp_params_data = tmp_params.host_data();
    cut_points_val.resize(tmp_params_data[0]);
    cut_row_ptr.resize(n_column + 1);
    cut_fid.resize(tmp_params_data[0]);

    cut_row_ptr.copy_from(tmp_row_ptr);
    auto cut_row_ptr_data = cut_row_ptr.host_data();
    tmp_row_ptr_data = tmp_row_ptr.host_data();
    for(int i = 1; i < (n_column + 1); i++) {
        if(tmp_row_ptr_data[i] <= tmp_params_data[1])
            cut_row_ptr_data[i] += cut_row_ptr_data[i-1];
        else
            cut_row_ptr_data[i] = cut_row_ptr_data[i-1] + max_num_bins;
    }

    auto cut_point_val_data = cut_points_val.device_data();
    tmp_row_ptr_data = tmp_row_ptr.device_data();
    tmp_params_data = tmp_params.device_data();
    cut_row_ptr_data = cut_row_ptr.device_data();
    unique_vals_data = unique_vals.device_data();
    device_loop_2d(n_column, cut_row_ptr.device_data(), [=] __device__ (int fid, int i){
        // fid --> [0, n_column)  &&  i -->> [cut_row_ptr[fid], cut_row_ptr[fid+1])
        int unique_len = tmp_row_ptr_data[fid+1];
        int unique_idx =  i - cut_row_ptr_data[fid];
        int cp_idx = (unique_len <= tmp_params_data[1]) ? unique_idx : (unique_len / tmp_params_data[1] * unique_idx);
        cut_point_val_data[i] = unique_vals_data[fid*n_instances + cp_idx];
    });

    auto cut_fid_data = cut_fid.device_data();
    device_loop_2d(n_column, cut_row_ptr.device_data(), [=] __device__(int fid, int i) {
        cut_fid_data[i] = fid;
    });
}

template<typename T>
void syncarray_resize(SyncArray<T> &buf_array, int new_size) {
    CHECK_GE(buf_array.size(), new_size) << "The size of the target Syncarray must greater than the new size. ";
    SyncArray<T> tmp_array(new_size);
    tmp_array.copy_from(buf_array.device_data(), new_size);
    buf_array.resize(new_size);
    buf_array.copy_from(tmp_array);
}

void unique_by_flag(SyncArray<float> &target_arr, SyncArray<int> &flags, int n_columns) {
    using namespace thrust::placeholders;

    float max_elem = max_elements(target_arr);
    LOG(DEBUG) << "max feature value: " << max_elem;
    CHECK_LT(max_elem + n_columns*(max_elem + 1),INT_MAX) << "Max_values is too large to be transformed";

    // 1. transform data into unique ranges
    thrust::transform(thrust::device,
                      target_arr.device_data(),
                      target_arr.device_end(),
                      flags.device_data(),
                      target_arr.device_data(),
                      (_1 + _2 * (max_elem + 1)));
    // 2. sort the transformed data
    sort_array(target_arr, false);
    thrust::reverse(thrust::device, flags.device_data(), flags.device_end());
    // 3. eliminate duplicates
    auto new_end = thrust::unique_by_key(thrust::device, target_arr.device_data(), target_arr.device_end(),
                                         flags.device_data());
    int new_size = new_end.first - target_arr.device_data();
    syncarray_resize(target_arr, new_size);
    syncarray_resize(flags, new_size);
    // 4. transform data back
    thrust::transform(thrust::device, target_arr.device_data(),
                      target_arr.device_end(),
                      flags.device_data(),
                      target_arr.device_data(),
                      (_1 - _2 * (max_elem + 1)));
    cub_sort_by_key(flags, target_arr);
}

/**
 * fast but cost more memory
 */
void HistCut::get_cut_points3(SparseColumns &columns, int max_num_bins, int n_instances) {
    LOG(INFO) << "Fast getting cut points...";
    int n_column = columns.n_column;

    cut_points_val.resize(columns.csc_val.size());
    cut_row_ptr.resize(columns.csc_col_ptr.size());
    cut_fid.resize(columns.csc_val.size());
    cut_points_val.copy_from(columns.csc_val);

    auto cut_fid_data = cut_fid.device_data();
    device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=] __device__(int fid, int i) {
        cut_fid_data[i] = fid;
    });
    unique_by_flag(cut_points_val, cut_fid, n_column);

    cut_row_ptr.resize(n_column + 1);
    auto cut_row_ptr_data = cut_row_ptr.device_data();
    device_loop(cut_fid.size(), [=] __device__(int fid) {
        atomicAdd(cut_row_ptr_data + cut_fid_data[fid] + 1, 1);
    });
    thrust::inclusive_scan(thrust::device, cut_row_ptr_data, cut_row_ptr_data + cut_row_ptr.size(), cut_row_ptr_data);

    SyncArray<int> select_index(cut_fid.size());
    auto select_index_data = select_index.device_data();
    device_loop_2d_with_maximum(n_column, cut_row_ptr_data, max_num_bins, [=] __device__(int fid, int i, int interval) {
        int feature_idx = i - cut_row_ptr_data[fid];
        if(interval == 0)
            select_index_data[i] = 1;
        else if(feature_idx < max_num_bins)
            select_index_data[cut_row_ptr_data[fid] + interval * feature_idx] = 1;
    });

    cub_select(cut_fid, select_index);
    cub_select(cut_points_val, select_index);

    cut_fid_data = cut_fid.device_data();
    cut_row_ptr.resize(n_column + 1);
    cut_row_ptr_data = cut_row_ptr.device_data();
    device_loop(cut_fid.size(), [=] __device__(int fid) {
        atomicAdd(cut_row_ptr_data + cut_fid_data[fid] + 1, 1);
    });
    thrust::inclusive_scan(thrust::device, cut_row_ptr_data, cut_row_ptr_data + cut_row_ptr.size(), cut_row_ptr_data);

    LOG(DEBUG) << "--->>>>  cut points value: " << cut_points_val;
    LOG(DEBUG) << "--->>>> cut row ptr: " << cut_row_ptr;
    LOG(DEBUG) << "--->>>> cut fid: " << cut_fid;
    LOG(DEBUG) << "TOTAL CP:" << cut_fid.size();
    LOG(DEBUG) << "NNZ: " << columns.csc_val.size();
}
