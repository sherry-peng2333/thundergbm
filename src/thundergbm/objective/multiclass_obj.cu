#include "hip/hip_runtime.h"
//
// Created by ss on 19-1-15.
//
#include "thundergbm/objective/multiclass_obj.h"

void
Softmax::get_gradient(const SyncArray<float_type> &y, const SyncArray<float_type> &y_p, SyncArray<GHPair> &gh_pair) {
    CHECK_EQ(y.size(), y_p.size() / num_class);
    CHECK_EQ(y_p.size(), gh_pair.size());
    auto y_data = y.device_data();
    auto yp_data = y_p.device_data();
    auto gh_data = gh_pair.device_data();
    int num_class = this->num_class;
    int n_instances = y_p.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        float_type max = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            max = fmaxf(max, yp_data[k * n_instances + i]);
        }
        float_type sum = 0;
        for (int k = 0; k < num_class; ++k) {
            //-max to avoid numerical issue
            sum += expf(yp_data[k * n_instances + i] - max);
        }
        for (int k = 0; k < num_class; ++k) {
            float_type p = expf(yp_data[k * n_instances + i] - max) / sum;
            //gradient = p_i - y_i
            //approximate hessian = 2 * p_i * (1 - p_i)
            //https://github.com/dmlc/xgboost/issues/2485
            float_type g = k == y_data[i] ? (p - 1) : (p - 0);
            float_type h = fmaxf(2 * p * (1 - p), 1e-16f);
            gh_data[k * n_instances + i] = GHPair(g, h);
        }
    });
}

void Softmax::configure(GBMParam param, const DataSet &dataset) {
    num_class = param.num_class;
    label.resize(num_class);
    CHECK_EQ(dataset.label.size(), num_class)<<dataset.label.size() << "!=" << num_class;
    label.copy_from(dataset.label.data(), num_class);
}

void Softmax::predict_transform(SyncArray<float_type> &y) {
    //this method transform y(#class * #instances) into y(#instances)
    auto yp_data = y.device_data();
    auto label_data = label.device_data();
    int num_class = this->num_class == 2 ? 1 : this->num_class;
    int n_instances = y.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        int max_k = 0;
        float_type max_p = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            if (max_p < yp_data[k * n_instances + i]) {
                max_p = yp_data[k * n_instances + i];
                max_k = k;
            }
        }
        yp_data[i] = label_data[max_k];
    });
    //TODO not to make a temp_y?
    SyncArray<float_type> temp_y(n_instances);
    temp_y.copy_from(y.device_data(), n_instances);
    y.resize(n_instances);
    y.copy_from(temp_y);
    LOG(INFO) << "Done y transform size is " << y.size();
}


void SoftmaxProb::predict_transform(SyncArray<float_type> &y) {
    auto yp_data = y.device_data();
    int num_class = this->num_class;
    int n_instances = y.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        float_type max = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            max = fmaxf(max, yp_data[k * n_instances + i]);
        }
        float_type sum = 0;
        for (int k = 0; k < num_class; ++k) {
            //-max to avoid numerical issue
            yp_data[k * n_instances + i] = expf(yp_data[k * n_instances + i] - max);
            sum += yp_data[k * n_instances + i];
        }
        for (int k = 0; k < num_class; ++k) {
            yp_data[k * n_instances + i] /= sum;
        }
    });
}

void MoClassObj::get_gradient(const SyncArray<float_type> &y, const SyncArray<float_type> &y_p, SyncArray<GHPair> &gh_pair){
    CHECK_EQ(y.size(), y_p.size() / d_outputs_);
    CHECK_EQ(y_p.size(), gh_pair.size());
    auto y_data = y.device_data();
    auto yp_data = y_p.device_data();
    auto gh_data = gh_pair.device_data();
    int d_outputs_ = this->d_outputs_;
    int n_instances = y_p.size() / d_outputs_;
    device_loop(n_instances, [=]__device__(int i) {
        float_type max = yp_data[i];
        for (int k = 1; k < d_outputs_; ++k) {
            max = fmaxf(max, yp_data[i * n_instances + k]);
        }
        float_type sum = 0;
        for (int k = 0; k < d_outputs_; ++k) {
            //-max to avoid numerical issue
            sum += expf(yp_data[i * n_instances + k] - max);
        }
        for (int k = 0; k < d_outputs_; ++k) {
            float_type p = expf(yp_data[i * n_instances + k] - max) / sum;
            //gradient = p_i - y_i
            //approximate hessian = 2 * p_i * (1 - p_i)
            //https://github.com/dmlc/xgboost/issues/2485
            float_type g = k == y_data[i] ? (p - 1) : (p - 0);
            float_type h = fmaxf(2 * p * (1 - p), 1e-16f);
            gh_data[i * n_instances + k] = GHPair(g, h);
        }
    });
}

void MoClassObj::configure(GBMParam param, const DataSet &dataset) {
    d_outputs_ = param.d_outputs_;
    label.resize(d_outputs_);
    CHECK_EQ(dataset.label.size(), d_outputs_)<<dataset.label.size() << "!=" << d_outputs_;
    label.copy_from(dataset.label.data(), d_outputs_);
}

void MoClassObj::predict_transform(SyncArray<float_type> &y) {
    //this method transform y(#class * #instances) into y(#instances)
    auto yp_data = y.device_data();
    auto label_data = label.device_data();
    int d_outputs_ = this->d_outputs_;
    int n_instances = y.size() / d_outputs_;
    device_loop(n_instances, [=]__device__(int i) {
        int max_k = 0;
        float_type max_p = yp_data[i];
        for (int k = 1; k < d_outputs_; ++k) {
            if (max_p < yp_data[i * n_instances + k]) {
                max_p = yp_data[i * n_instances + k];
                max_k = k;
            }
        }
        yp_data[i] = label_data[max_k];
    });
    //TODO not to make a temp_y?
    SyncArray<float_type> temp_y(n_instances);
    temp_y.copy_from(y.device_data(), n_instances);
    y.resize(n_instances);
    y.copy_from(temp_y);
    LOG(INFO) << "Done y transform size is " << y.size();
}