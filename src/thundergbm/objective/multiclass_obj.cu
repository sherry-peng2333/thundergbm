#include "hip/hip_runtime.h"
//
// Created by ss on 19-1-15.
//
#include "thundergbm/objective/multiclass_obj.h"

void Softmax::get_gradient(const SyncArray<float_type> &y, const SyncArray<float_type> &y_p, SyncArray<GHPair> &gh_pair) {
    CHECK_EQ(y.size(), y_p.size() / num_class);
    CHECK_EQ(y_p.size(), gh_pair.size());
    auto y_data = y.device_data();
    auto yp_data = y_p.device_data();
    auto gh_data = gh_pair.device_data();
    int num_class = this->num_class;
    int n_instances = y_p.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        float_type max = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            max = fmaxf(max, yp_data[k * n_instances + i]);
        }
        float_type sum = 0;
        for (int k = 0; k < num_class; ++k) {
            //-max to avoid numerical issue
            sum += expf(yp_data[k * n_instances + i] - max);
        }
        for (int k = 0; k < num_class; ++k) {
            float_type p = expf(yp_data[k * n_instances + i] - max) / sum;
            //gradient = p_i - y_i
            //approximate hessian = 2 * p_i * (1 - p_i)
            //https://github.com/dmlc/xgboost/issues/2485
            float_type g = k == y_data[i] ? (p - 1) : (p - 0);
            float_type h = fmaxf(2 * p * (1 - p), 1e-16f);
            gh_data[k * n_instances + i] = GHPair(g, h);
        }
    });
}

void Softmax::configure(GBMParam param, const DataSet &dataset) {
    num_class = param.num_class;
    label.resize(num_class);
    CHECK_EQ(dataset.label.size(), num_class)<<dataset.label.size() << "!=" << num_class;
    label.copy_from(dataset.label.data(), num_class);
}

void Softmax::predict_transform(SyncArray<float_type> &y) {
    //this method transform y(#class * #instances) into y(#instances)
    auto yp_data = y.device_data();
    auto label_data = label.device_data();
    int num_class = this->num_class == 2 ? 1 : this->num_class;
    int n_instances = y.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        int max_k = 0;
        float_type max_p = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            if (max_p < yp_data[k * n_instances + i]) {
                max_p = yp_data[k * n_instances + i];
                max_k = k;
            }
        }
        yp_data[i] = label_data[max_k];
    });
    //TODO not to make a temp_y?
    SyncArray<float_type> temp_y(n_instances);
    temp_y.copy_from(y.device_data(), n_instances);
    y.resize(n_instances);
    y.copy_from(temp_y);
    LOG(INFO) << "Done y transform size is " << y.size();
}


void SoftmaxProb::predict_transform(SyncArray<float_type> &y) {
    auto yp_data = y.device_data();
    int num_class = this->num_class;
    int n_instances = y.size() / num_class;
    device_loop(n_instances, [=]__device__(int i) {
        float_type max = yp_data[i];
        for (int k = 1; k < num_class; ++k) {
            max = fmaxf(max, yp_data[k * n_instances + i]);
        }
        float_type sum = 0;
        for (int k = 0; k < num_class; ++k) {
            //-max to avoid numerical issue
            yp_data[k * n_instances + i] = expf(yp_data[k * n_instances + i] - max);
            sum += yp_data[k * n_instances + i];
        }
        for (int k = 0; k < num_class; ++k) {
            yp_data[k * n_instances + i] /= sum;
        }
    });
}
