//
// Created by zeyi on 1/9/19.
//
#include <fstream>
#include "hip/hip_runtime_api.h"

#include <thundergbm/tree.h>
#include <thundergbm/trainer.h>
#include <thundergbm/metric/metric.h>
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/reduce.h"
#include "time.h"
#include "thundergbm/booster.h"
#include "chrono"
#include <thundergbm/parser.h>
using namespace std;

vector<vector<Tree>> TreeTrainer::train(GBMParam &param, const DataSet &dataset) {
    if (param.tree_method == "auto")
        if (dataset.n_features() > 20000)
            param.tree_method = "exact";
        else
            param.tree_method = "hist";

    //correct the number of classes
    if(param.objective.find("multi:") != std::string::npos || param.objective.find("binary:") != std::string::npos) {
        int num_class = dataset.label.size();
        if (param.num_class != num_class) {
            LOG(INFO) << "updating number of classes from " << param.num_class << " to " << num_class;
            param.num_class = num_class;
        }
        if(param.num_class > 2)
            param.tree_per_rounds = param.num_class;
    }
    else{
        param.num_class = 1;
        if(param.objective.find("mo-lab:") != std::string::npos || param.objective.find("mo-reg:") != std::string::npos){
            param.d_outputs = dataset.d_outputs_;
        }
    }
//    else if(param.objective.find("mo-lab:") != std::string::npos || param.objective.find("mo-reg:") != std::string::npos){
//        int num_class = dataset.d_outputs_;
//        if (param.num_class != num_class) {
//            LOG(INFO) << "updating number of classes from " << param.num_class << " to " << num_class;
//            param.num_class = num_class;
//        }
//    }
//    else if(param.objective.find("reg:") != std::string::npos){
//        param.num_class = 1;
//    }

    vector<vector<Tree>> boosted_model;
    Booster booster;
    booster.init(dataset, param);
    std::chrono::high_resolution_clock timer;
    auto start = timer.now();
    for (int i = 0; i < param.n_trees; ++i) {
        //one iteration may produce multiple trees, depending on objectives
        booster.boost(boosted_model);
    }
    auto stop = timer.now();
    std::chrono::duration<float> training_time = stop - start;
    LOG(INFO) << "training time = " << training_time.count();

    std::atexit([]() {
        SyncMem::clear_cache();
    });
	// SyncMem::clear_cache();
	return boosted_model;
}
