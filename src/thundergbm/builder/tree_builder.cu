//
// Created by jiashuai on 19-1-23.
//

#include <thundergbm/builder/tree_builder.h>
#include "thundergbm/util/multi_device.h"
#include "thundergbm/util/device_lambda.cuh"

void TreeBuilder::update_tree() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto& sp = this->sp[device_id];
        auto& tree = this->trees[device_id];
        auto sp_data = sp.device_data();
        LOG(DEBUG) << sp;
        int n_nodes_in_level = sp.size();

        Tree::TreeNode *nodes_data = tree.nodes.device_data();
        float_type rt_eps = param.rt_eps;
        float_type lambda = param.lambda;

        device_loop(n_nodes_in_level, [=]__device__(int i) {
            float_type best_split_gain = sp_data[i].gain;
            if (best_split_gain > rt_eps) {
                //do split
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.gain = best_split_gain;

                Tree::TreeNode &lch = nodes_data[node.lch_index];//left child
                Tree::TreeNode &rch = nodes_data[node.rch_index];//right child
                lch.is_valid = true;
                rch.is_valid = true;
                node.split_feature_id = sp_data[i].split_fea_id;
                GHPair p_missing_gh = sp_data[i].fea_missing_gh;
                //todo process begin
                node.split_value = sp_data[i].fval;
                node.split_bid = sp_data[i].split_bid;
                rch.sum_gh_pair = sp_data[i].rch_sum_gh;
                if (sp_data[i].default_right) {
                    rch.sum_gh_pair = rch.sum_gh_pair + p_missing_gh;
                    node.default_right = true;
                }
                lch.sum_gh_pair = node.sum_gh_pair - rch.sum_gh_pair;
                lch.calc_weight(lambda);
                rch.calc_weight(lambda);
            } else {
                //set leaf
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.is_leaf = true;
                nodes_data[node.lch_index].is_valid = false;
                nodes_data[node.rch_index].is_valid = false;
            }
        });
        LOG(DEBUG) << tree.nodes;
    });
}

void TreeBuilder::predict_in_training(int k) {
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto y_predict_data = y_predict[device_id].device_data() + k * n_instances;
        auto nid_data = ins2node_id[device_id].device_data();
        const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
        auto lr = param.learning_rate;
        device_loop(n_instances, [=]__device__(int i) {
            int nid = nid_data[i];
            while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
            y_predict_data[i] += lr * nodes_data[nid].base_weight;
        });
    });
}

void TreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    int n_available_device;
    hipGetDeviceCount(&n_available_device);
    CHECK_GE(n_available_device, param.n_device) << "only " << n_available_device
                                                 << " GPUs available; please set correct number of GPUs to use";
    FunctionBuilder::init(dataset, param);
    this->n_instances = dataset.n_instances();
    trees = vector<Tree>(param.n_device);
    ins2node_id = MSyncArray<int>(param.n_device, n_instances);
    sp = MSyncArray<SplitPoint>(param.n_device);
    has_split = vector<bool>(param.n_device);
    int n_outputs = param.num_class * n_instances;
    y_predict = MSyncArray<float_type>(param.n_device, n_outputs);
    gradients = MSyncArray<GHPair>(param.n_device, n_instances);
    if(param.multi_outputs){
        sp_fea_missing_gh = MSyncArray<GHPair>(param.n_device);
        sp_rch_sum_gh = MSyncArray<GHPair>(param.n_device);
    }
}

void TreeBuilder::ins2node_id_all_reduce(int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = ins2node_id.front().device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(ins2node_id[d]);
            device_loop(n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        ins2node_id[device_id].copy_from(ins2node_id.front());
    });
}

void TreeBuilder::split_point_all_reduce(int depth) {
    TIMED_FUNC(timerObj);
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = sp.front().host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = sp[device_id].host_data();
        for (int j = 0; j < sp[device_id].size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos])
                global_sp_data[global_pos] = local_sp_data[j];
            else
                global_sp_data[global_pos] = (global_sp_data[global_pos].gain >= local_sp_data[j].gain)
                                             ?
                                             global_sp_data[global_pos] : local_sp_data[j];
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
       sp[device_id].copy_from(sp.front());
    });
    LOG(DEBUG) << "global best split point = " << sp.front();
}

vector<Tree> TreeBuilder::build_approximate(const MSyncArray<GHPair> &gradients) {
    vector<Tree> trees(param.tree_per_rounds);
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        this->shards[device_id].column_sampling(param.column_sampling_rate);
    });

    for (int k = 0; k < param.tree_per_rounds; ++k) {
        Tree &tree = trees[k];
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->ins2node_id[device_id].resize(n_instances);
            this->gradients[device_id].set_device_data(const_cast<GHPair *>(gradients[device_id].device_data() + k * n_instances));
            this->trees[device_id].init2(this->gradients[device_id], param);
        });

        for (int level = 0; level < param.depth; ++level) {
            DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
                find_split(level, device_id);
            });
            split_point_all_reduce(level);
            {
                TIMED_SCOPE(timerObj, "apply sp");
                update_tree();
                update_ins2node_id();
                {
                    LOG(TRACE) << "gathering ins2node id";
                    //get final result of the reset instance id to node id
                    bool has_split = false;
                    for (int d = 0; d < param.n_device; d++) {
                        has_split |= this->has_split[d];
                    }
                    if (!has_split) {
                        LOG(INFO) << "no splittable nodes, stop";
                        break;
                    }
                }
                ins2node_id_all_reduce(level);
            }
        }
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->trees[device_id].prune_self(param.gamma);
        });
        predict_in_training(k);
        tree.nodes.resize(this->trees.front().nodes.size());
        tree.nodes.copy_from(this->trees.front().nodes);
    }
    return trees;
}
