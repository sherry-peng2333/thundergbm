//
// Created by jiashuai on 19-1-23.
//

#include <thundergbm/builder/tree_builder.h>
#include "thundergbm/util/multi_device.h"
#include "thundergbm/util/device_lambda.cuh"

void TreeBuilder::update_tree() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto& sp = this->sp[device_id];
        auto& tree = this->trees[device_id];
        auto sp_data = sp.host_data();
        LOG(DEBUG) << sp;
        int n_nodes_in_level = sp.size();

        Tree::TreeNode *nodes_data = tree.nodes.host_data();
        float_type rt_eps = param.rt_eps;
        float_type lambda = param.lambda;

        for(int i = 0; i < n_nodes_in_level; i++){
            float_type best_split_gain = sp_data[i].gain;
            if (best_split_gain > rt_eps) {
                //do split
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.gain = best_split_gain;

                Tree::TreeNode &lch = nodes_data[node.lch_index];//left child
                Tree::TreeNode &rch = nodes_data[node.rch_index];//right child
                lch.is_valid = true;
                rch.is_valid = true;
                lch.base_weight = SyncArray<float_type>(d_outputs_);
                rch.base_weight = SyncArray<float_type>(d_outputs_);
                lch.sum_gh_pair = SyncArray<GHPair>(d_outputs_);
                rch.sum_gh_pair = SyncArray<GHPair>(d_outputs_);
                node.split_feature_id = sp_data[i].split_fea_id;
                auto p_missing_gh = sp_data[i].fea_missing_gh.device_data();
                //todo process begin
                node.split_value = sp_data[i].fval;
                node.split_bid = sp_data[i].split_bid;
                rch.sum_gh_pair.copy_from(sp_data[i].rch_sum_gh);
                auto lsum_gh_pair_data = lch.sum_gh_pair.device_data();
                auto rsum_gh_pair_data = rch.sum_gh_pair.device_data();
                auto nsum_gh_pair_data = node.sum_gh_pair.device_data();
                if (sp_data[i].default_right) {
                    device_loop(d_outputs_, [=]__device__(int j){
                        rsum_gh_pair_data[j] = rsum_gh_pair_data[j] + p_missing_gh[j];
                    });  
                    node.default_right = true;
                }
                device_loop(d_outputs_, [=]__device__(int j){
                    lsum_gh_pair_data[j] = nsum_gh_pair_data[j] - rsum_gh_pair_data[j];
                });
                lch.calc_weight(lambda, d_outputs_);
                rch.calc_weight(lambda, d_outputs_);
            } else {
                //set leaf
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.is_leaf = true;
                nodes_data[node.lch_index].is_valid = false;
                nodes_data[node.rch_index].is_valid = false;
            }
        }
        LOG(DEBUG) << tree.nodes;
    });
}

void TreeBuilder::predict_in_training(int k) {
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto y_predict_data = y_predict[device_id].device_data() + k * n_instances * d_outputs_;
        auto nid_data = ins2node_id[device_id].host_data();
        const Tree::TreeNode *nodes_data = trees[device_id].nodes.host_data();
        auto lr = param.learning_rate;
        int d_outputs_ = this->d_outputs_;
        for(int i = 0; i < n_instances; i++){
            int nid = nid_data[i];
            while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
            auto lr_nodes_data = nodes_data[nid].base_weight.device_data();
            device_loop(d_outputs_, [=]__device__(int j){
                y_predict_data[i * d_outputs_ + j] += lr * lr_nodes_data[j];
            });
        }
    });
}

void TreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    int n_available_device;
    hipGetDeviceCount(&n_available_device);
    CHECK_GE(n_available_device, param.n_device) << "only " << n_available_device
                                                 << " GPUs available; please set correct number of GPUs to use";
    FunctionBuilder::init(dataset, param);      // this->param = param
    this->n_instances = dataset.n_instances();
    this->d_outputs_ = dataset.d_outputs_;
    trees = vector<Tree>(param.n_device);
    ins2node_id = MSyncArray<int>(param.n_device, n_instances);
    sp = MSyncArray<SplitPoint>(param.n_device);
    has_split = vector<bool>(param.n_device);
    int n_outputs = param.num_class * n_instances;
    y_predict = MSyncArray<float_type>(param.n_device, n_outputs);
    gradients = MSyncArray<GHPair>(param.n_device, n_instances*this->d_outputs_);
}

void TreeBuilder::ins2node_id_all_reduce(int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = ins2node_id.front().device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(ins2node_id[d]);
            device_loop(n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        ins2node_id[device_id].copy_from(ins2node_id.front());
    });
}

void TreeBuilder::split_point_all_reduce(int depth) {
    TIMED_FUNC(timerObj);
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = sp.front().host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = sp[device_id].host_data();
        for (int j = 0; j < sp[device_id].size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos]){
                global_sp_data[global_pos].gain = local_sp_data[j].gain;
                global_sp_data[global_pos].default_right = local_sp_data[j].default_right;
                global_sp_data[global_pos].nid = local_sp_data[j].nid;
                global_sp_data[global_pos].split_fea_id = local_sp_data[j].split_fea_id;
                global_sp_data[global_pos].fval = local_sp_data[j].fval;
                global_sp_data[global_pos].split_bid = local_sp_data[j].split_bid;
                global_sp_data[global_pos].fea_missing_gh.copy_from(local_sp_data[j].fea_missing_gh);
                global_sp_data[global_pos].rch_sum_gh.copy_from(local_sp_data[j].rch_sum_gh);}
            else if(global_sp_data[global_pos].gain < local_sp_data[j].gain){
                global_sp_data[global_pos].gain = local_sp_data[j].gain;
                global_sp_data[global_pos].default_right = local_sp_data[j].default_right;
                global_sp_data[global_pos].nid = local_sp_data[j].nid;
                global_sp_data[global_pos].split_fea_id = local_sp_data[j].split_fea_id;
                global_sp_data[global_pos].fval = local_sp_data[j].fval;
                global_sp_data[global_pos].split_bid = local_sp_data[j].split_bid;
                global_sp_data[global_pos].fea_missing_gh.copy_from(local_sp_data[j].fea_missing_gh);
                global_sp_data[global_pos].rch_sum_gh.copy_from(local_sp_data[j].rch_sum_gh);}
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
       sp[device_id].copy_from(sp.front());
    });
    LOG(DEBUG) << "global best split point = " << sp.front();
}

vector<Tree> TreeBuilder::build_approximate(const MSyncArray<GHPair> &gradients) {
    vector<Tree> trees(param.tree_per_rounds);
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        this->shards[device_id].column_sampling(param.column_sampling_rate);
    });
    for (int k = 0; k < param.tree_per_rounds; ++k) {
        Tree &tree = trees[k];
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->ins2node_id[device_id].resize(n_instances);
            this->gradients[device_id].set_device_data(const_cast<GHPair *>(gradients[device_id].device_data() + k * n_instances * d_outputs_));
            this->trees[device_id].init2(this->gradients[device_id], param, this->d_outputs_);
        });
        for (int level = 0; level < param.depth; ++level) {
            DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
                find_split(level, device_id);
            });
            split_point_all_reduce(level);
            //LOG(INFO) << "split_point_all_reduce";
            {
                TIMED_SCOPE(timerObj, "apply sp");
                update_tree();
                //LOG(INFO) << "update_tree";
                update_ins2node_id();
                //LOG(INFO) << "update_ins2node_id";
                {
                    LOG(TRACE) << "gathering ins2node id";
                    //get final result of the reset instance id to node id
                    bool has_split = false;
                    for (int d = 0; d < param.n_device; d++) {
                        has_split |= this->has_split[d];
                    }
                    if (!has_split) {
                        LOG(INFO) << "no splittable nodes, stop";
                        break;
                    }
                }
                ins2node_id_all_reduce(level);
            }
        }
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->trees[device_id].prune_self(param.gamma);
        });
        predict_in_training(k);
        tree.nodes.resize(this->trees.front().nodes.size());
        tree.nodes.copy_from(this->trees.front().nodes);
        string s = tree.dump(param.depth);
        LOG(INFO) << "TREE:" << s;

    }
    LOG(INFO) << "one tree............";
    return trees;
}
