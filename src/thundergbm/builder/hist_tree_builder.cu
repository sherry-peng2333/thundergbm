#include "hip/hip_runtime.h"
//
// Created by ss on 19-1-20.
//
#include "thundergbm/builder/hist_tree_builder.h"

#include "thundergbm/util/cub_wrapper.h"
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/iterator/counting_iterator.h"
#include "thrust/iterator/transform_iterator.h"
#include "thrust/iterator/discard_iterator.h"
#include "thrust/sequence.h"
#include "thrust/binary_search.h"
#include "thundergbm/util/multi_device.h"

void HistTreeBuilder::get_bin_ids() {
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        SparseColumns &columns = shards[device_id].columns;
        HistCut &cut = this->cut[device_id];
        auto &dense_bin_id = this->dense_bin_id[device_id];
        using namespace thrust;
        int n_column = columns.n_column;
        int nnz = columns.nnz;
        auto cut_row_ptr = cut.cut_row_ptr.device_data();
        auto cut_points_ptr = cut.cut_points_val.device_data();
        auto csc_val_data = columns.csc_val.device_data();
        SyncArray<unsigned char> bin_id;
        bin_id.resize(columns.nnz);
        auto bin_id_data = bin_id.device_data();
        int n_block = fminf((nnz / n_column - 1) / 256 + 1, 4 * 56);
        {
            auto lowerBound = [=]__device__(const float_type *search_begin, const float_type *search_end, float_type val) {
                const float_type *left = search_begin;
                const float_type *right = search_end - 1;

                while (left != right) {
                    const float_type *mid = left + (right - left) / 2;
                    if (*mid <= val)
                        right = mid;
                    else left = mid + 1;
                }
                return left;
            };
            TIMED_SCOPE(timerObj, "binning");
            device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int cid, int i) {
                auto search_begin = cut_points_ptr + cut_row_ptr[cid];
                auto search_end = cut_points_ptr + cut_row_ptr[cid + 1];
                auto val = csc_val_data[i];
                bin_id_data[i] = lowerBound(search_begin, search_end, val) - search_begin;
            }, n_block);
        }

        auto max_num_bin = param.max_num_bin;
        dense_bin_id.resize(n_instances * n_column);
        auto dense_bin_id_data = dense_bin_id.device_data();
        auto csc_row_idx_data = columns.csc_row_idx.device_data();
        device_loop(n_instances * n_column, [=]__device__(int i) {
        dense_bin_id_data[i] = max_num_bin;
    });
        device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int fid, int i) {
        int row = csc_row_idx_data[i];
        unsigned char bid = bin_id_data[i];
        dense_bin_id_data[row * n_column + fid] = bid;
    }, n_block);
    });
}

void HistTreeBuilder::find_split(int level, int device_id) {
    std::chrono::high_resolution_clock timer;

    const SparseColumns &columns = shards[device_id].columns;
    SyncArray<int> &nid = ins2node_id[device_id];
    SyncArray<GHPair> &gh_pair = gradients[device_id];
    Tree &tree = trees[device_id];
    SyncArray<SplitPoint> &sp = this->sp[device_id];
    SyncArray<bool> &ignored_set = shards[device_id].ignored_set;
    HistCut &cut = this->cut[device_id];
    auto &dense_bin_id = this->dense_bin_id[device_id];
    auto &last_hist = this->last_hist[device_id];

    TIMED_FUNC(timerObj);
    int n_nodes_in_level = static_cast<int>(pow(2, level));
    int nid_offset = static_cast<int>(pow(2, level) - 1);
    int n_column = columns.n_column;
    int n_partition = n_column * n_nodes_in_level;
    int n_bins = cut.cut_points_val.size();
    int n_max_nodes = 2 << param.depth;
    int n_max_splits = n_max_nodes * n_bins;
    int n_split = n_nodes_in_level * n_bins;

    LOG(TRACE) << "start finding split";

    //find the best split locally
    {
        using namespace thrust;
        auto t_build_start = timer.now();

        //calculate split information for each split
        SyncArray<GHPair> hist(n_max_splits*d_outputs_);
        SyncArray<GHPair> missing_gh(n_partition*d_outputs_);
        auto cut_fid_data = cut.cut_fid.device_data();
        auto i2fid = [=] __device__(int i) { return cut_fid_data[i % n_bins]; };
        auto hist_fid = make_transform_iterator(counting_iterator<int>(0), i2fid);
        {
            {
                TIMED_SCOPE(timerObj, "build hist");
                {
                    size_t
                    smem_size = n_bins * sizeof(GHPair)*d_outputs_;
                    LOG(DEBUG) << "shared memory size = " << smem_size / 1024.0 << " KB";
                    if (n_nodes_in_level == 1) {
                        //root
                        auto hist_data = hist.device_data();
                        auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                        auto gh_data = gh_pair.device_data();
                        auto dense_bin_id_data = dense_bin_id.device_data();
                        auto max_num_bin = param.max_num_bin;
                        auto n_instances = this->n_instances;
                        if (smem_size > 600) {
                            //48 * 1024
                            device_loop(n_instances * n_column, [=]__device__(int i) {
                                int iid = i / n_column;
                                int fid = i % n_column;
                                unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                if (bid != max_num_bin) {
                                    int feature_offset = cut_row_ptr_data[fid];
                                    for(int i = 0; i < d_outputs_; i++){
                                        const GHPair src = gh_data[iid*d_outputs_+i];
                                        GHPair &dest = hist_data[i*n_max_splits+feature_offset + bid];
                                        if(src.h != 0)
                                            atomicAdd(&dest.h, src.h);
                                        if(src.g != 0)
                                            atomicAdd(&dest.g, src.g);
                                    }

                                }
                            });
                        } else {
                            // To do: add multi-outputs feature 
                            int num_fv = n_instances * n_column;
                            anonymous_kernel([=]__device__() {
                                extern __shared__ GHPair local_hist[];
                                for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                    local_hist[i] = 0;
                                }
                                __syncthreads();
                                for (int i = blockIdx.x * blockDim.x + threadIdx.x;
                                     i < num_fv; i += blockDim.x * gridDim.x) {
                                    int iid = i / n_column;
                                    int fid = i % n_column;
                                    unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                    if (bid != max_num_bin) {
                                        int feature_offset = cut_row_ptr_data[fid];
                                        const GHPair src = gh_data[iid];
                                        GHPair &dest = local_hist[feature_offset + bid];
                                        if(src.h != 0)
                                            atomicAdd(&dest.h, src.h);
                                        if(src.g != 0)
                                            atomicAdd(&dest.g, src.g);

                                    }
                                }
                                __syncthreads();
                                for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                    GHPair &dest = hist_data[i];
                                    GHPair src = local_hist[i];
                                    if(src.h != 0)
                                        atomicAdd(&dest.h, src.h);
                                    if(src.g != 0)
                                        atomicAdd(&dest.g, src.g);

                                }
                            }, num_fv, smem_size);
                        }
                    } else {
                        //otherwise
                        auto t_dp_begin = timer.now();
                        SyncArray<int> node_idx(n_instances);
                        SyncArray<int> node_ptr(n_nodes_in_level + 1);
                        {
                            TIMED_SCOPE(timerObj, "data partitioning");
                            SyncArray<int> nid4sort(n_instances);
                            nid4sort.copy_from(ins2node_id[device_id]);
                            sequence(cuda::par, node_idx.device_data(), node_idx.device_end(), 0);
                            cub_sort_by_key(nid4sort, node_idx);
                            auto counting_iter = make_counting_iterator < int > (nid_offset);
                            node_ptr.host_data()[0] =
                                    lower_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), nid_offset) -
                                    nid4sort.device_data();

                            upper_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), counting_iter,
                                        counting_iter + n_nodes_in_level, node_ptr.device_data() + 1);
                            LOG(DEBUG) << "node ptr = " << node_ptr;
                            hipDeviceSynchronize();
                        }
                        auto t_dp_end = timer.now();
                        std::chrono::duration<double> dp_used_time = t_dp_end - t_dp_begin;
                        this->total_dp_time += dp_used_time.count();


                        auto node_ptr_data = node_ptr.host_data();
                        auto node_idx_data = node_idx.device_data();
                        auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                        auto gh_data = gh_pair.device_data();
                        auto dense_bin_id_data = dense_bin_id.device_data();
                        auto max_num_bin = param.max_num_bin;
                        for (int i = 0; i < n_nodes_in_level / 2; ++i) {

                            int nid0_to_compute = i * 2;
                            int nid0_to_substract = i * 2 + 1;
                            int n_ins_left = node_ptr_data[nid0_to_compute + 1] - node_ptr_data[nid0_to_compute];
                            int n_ins_right = node_ptr_data[nid0_to_substract + 1] - node_ptr_data[nid0_to_substract];
                            if (max(n_ins_left, n_ins_right) == 0) continue;
                            if (n_ins_left > n_ins_right)
                                swap(nid0_to_compute, nid0_to_substract);

                            //compute
                            {
                                int nid0 = nid0_to_compute;
                                auto idx_begin = node_ptr.host_data()[nid0];
                                auto idx_end = node_ptr.host_data()[nid0 + 1];
                                auto hist_data = hist.device_data();
                                //auto hist_data = hist.device_data() + nid0 * n_bins * d_outputs_;
                                this->total_hist_num++;

                                if (smem_size > 600) {
                                    //48 * 1024
                                    device_loop((idx_end - idx_begin) * n_column, [=]__device__(int i) {
                                        int iid = node_idx_data[i / n_column + idx_begin];
                                        int fid = i % n_column;
                                        unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                        if (bid != max_num_bin) {
                                            int feature_offset = cut_row_ptr_data[fid];
                                            for(int i = 0; i < d_outputs_; i++){
                                                const GHPair src = gh_data[iid*d_outputs_+i];
                                                GHPair &dest = hist_data[i*n_max_splits+nid0*n_bins+feature_offset + bid];
                                                if(src.h != 0)
                                                    atomicAdd(&dest.h, src.h);
                                                if(src.g != 0)
                                                    atomicAdd(&dest.g, src.g);
                                            }
                                        }
                                    });
                                } else {
                                    // To do: add multi-outputs feature
                                    int num_fv = (idx_end - idx_begin) * n_column;
                                    anonymous_kernel([=] __device__() {
                                        extern __shared__ GHPair local_hist[];
                                        for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                            local_hist[i] = 0;
                                        }
                                        __syncthreads();

                                        for (int i = blockIdx.x * blockDim.x + threadIdx.x;
                                             i < num_fv; i += blockDim.x * gridDim.x) {
                                            int iid = node_idx_data[i / n_column + idx_begin];
                                            //int fid = i - n_column *( i / n_column);
                                            int fid = i % n_column;
                                            unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                            if (bid != max_num_bin) {
                                                const GHPair src = gh_data[iid];
                                                int feature_offset = cut_row_ptr_data[fid];
                                                GHPair &dest = local_hist[feature_offset + bid];
                                                if(src.h != 0)
                                                    atomicAdd(&dest.h, src.h);
                                                if(src.g != 0)
                                                    atomicAdd(&dest.g, src.g);
                                            }
                                        }

                                        __syncthreads();
                                        for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                            GHPair src = local_hist[i];
                                            GHPair &dest = hist_data[i];
                                            if(src.h != 0)
                                                atomicAdd(&dest.h, src.h);
                                            if(src.g != 0)
                                                atomicAdd(&dest.g, src.g);
                                        }
                                    }, num_fv, smem_size);
                                }
                            }

                            //subtract
                            auto t_copy_start = timer.now();
                            {
                                auto hist_data_computed = hist.device_data() + nid0_to_compute * n_bins;
                                auto hist_data_to_compute = hist.device_data() + nid0_to_substract * n_bins;
                                auto father_hist_data = last_hist.device_data() + (nid0_to_substract / 2) * n_bins ;
                                //auto hist_data = hist.device_data();
                                //auto father_hist_data = last_hist.device_data();
                                device_loop(n_bins * d_outputs_, [=]__device__(int i) {
                                    int d = i / n_bins;
                                    int bid = i % n_bins;
                                    hist_data_computed[d*n_max_splits+bid] = father_hist_data[d*n_max_splits+bid]-hist_data_computed[d*n_max_splits+bid];
                                });
                            }
                            auto t_copy_end = timer.now();
                            std::chrono::duration<double> cp_used_time = t_copy_end - t_copy_start;
                            this->total_copy_time += cp_used_time.count();
//                            PERFORMANCE_CHECKPOINT(timerObj);
                        }  // end for each node
                    }//end # node > 1
                    last_hist.copy_from(hist);
                    hipDeviceSynchronize();
                }
                LOG(DEBUG) << "level: " << level;
                LOG(DEBUG) << "hist new = " << hist;
                auto t_build_hist_end = timer.now();
                std::chrono::duration<double> bh_used_time = t_build_hist_end - t_build_start;
                this->build_hist_used_time += bh_used_time.count();
                this->build_n_hist++;
                LOG(DEBUG) << "-------------->>> build_hist_used_time: " << bh_used_time.count();
                LOG(DEBUG) << "-------------->>> build_num_hist: " << this->build_n_hist;
                LOG(DEBUG) << "-------------->>> total_build_hist_used_time: " << this->build_hist_used_time - this->total_dp_time;
                LOG(DEBUG) << "-------------->>> n_hist::::: " << this->total_hist_num;
                LOG(DEBUG) << "-------------->>> dp_time::::: " << this->total_dp_time;
                LOG(DEBUG) << "-------------->>> cp_time::::: " << this->total_copy_time;

                //LOG(DEBUG) << "cutfid = " << cut.cut_fid;
                inclusive_scan_by_key(cuda::par, hist_fid, hist_fid + n_split*d_outputs_,
                                      hist.device_data(), hist.device_data());
                LOG(DEBUG) << hist;

                auto nodes_data = tree.nodes.device_data();
                auto missing_gh_data = missing_gh.device_data();
                auto cut_row_ptr = cut.cut_row_ptr.device_data();
                auto hist_data = hist.device_data();

                for(int pid = 0; pid < n_partition; pid++){
                    int nid0 = pid / n_column;
                    int nid = nid0 + nid_offset;
                    LOG(INFO) << "checkpoint ...";
                    if (!nodes_data[nid].splittable()) return;
                    int fid = pid % n_column;
                    auto sum_gh_pair_data = nodes_data[nid].sum_gh_pair.device_data();
                    if (cut_row_ptr[fid + 1] != cut_row_ptr[fid]){
                        device_loop(d_outputs_, [=]__device__(int i){
                            GHPair node_gh = hist_data[i*n_partition+nid0 * n_bins+cut_row_ptr[fid + 1] - 1];
                            missing_gh_data[i*n_partition+pid] = sum_gh_pair_data[i] - node_gh;
                        });
                    }
                }
                LOG(DEBUG) << missing_gh;
            }
        }
        //calculate gain of each split
        SyncArray<float_type> gain(n_max_splits);
        {
//            TIMED_SCOPE(timerObj, "calculate gain");
            auto compute_gain = [](GHPair father, GHPair lch, GHPair rch, float_type min_child_weight, float_type lambda) -> float {
                    if (lch.h >= min_child_weight && rch.h >= min_child_weight)
                    return (lch.g * lch.g) / (lch.h + lambda) + (rch.g * rch.g) / (rch.h + lambda) -
            (father.g * father.g) / (father.h + lambda);
                    else
                    return 0;
            };

            const Tree::TreeNode *nodes_data = tree.nodes.host_data();
            GHPair *gh_prefix_sum_data = hist.host_data();
            float_type *gain_data = gain.host_data();
            const auto missing_gh_data = missing_gh.host_data();
            auto ignored_set_data = ignored_set.host_data();
            //for lambda expression
            float_type mcw = param.min_child_weight;
            float_type l = param.lambda;

            for(int i = 0; i < n_split; i++){
                int nid0 = i / n_bins;
                int nid = nid0 + nid_offset;
                int fid = hist_fid[i % n_bins];
                if (nodes_data[nid].is_valid && !ignored_set_data[fid]) {
                    int pid = nid0 * n_column + hist_fid[i];
                    auto father_gh = nodes_data[nid].sum_gh_pair.host_data();
                    float_type default_to_left_gain = 0;
                    float_type default_to_right_gain = 0;
                    for(int j = 0; j < d_outputs_; j++){
                        GHPair rch_gh = gh_prefix_sum_data[j*n_max_splits+i];
                        default_to_left_gain = default_to_left_gain + max(0.f,
                            compute_gain(father_gh[j], father_gh[j] - rch_gh, rch_gh, mcw, l));
                    }
                    for(int j = 0; j < d_outputs_; j++){
                        GHPair rch_gh = gh_prefix_sum_data[j*n_max_splits+i]+missing_gh_data[j*n_partition+pid];
                        default_to_right_gain = max(0.f,
                            compute_gain(father_gh[j], father_gh[j] - rch_gh, rch_gh, mcw, l));
                    }
                    if (default_to_left_gain > default_to_right_gain)
                        gain_data[i] = default_to_left_gain;
                    else
                        gain_data[i] = -default_to_right_gain;//negative means default split to right

                } else gain_data[i] = 0;
            }

            LOG(DEBUG) << "gain = " << gain;
        }

        SyncArray<int_float> best_idx_gain(n_nodes_in_level);
        {
//            TIMED_SCOPE(timerObj, "get best gain");
            auto arg_abs_max = []__device__(const int_float &a, const int_float &b) {
                if (fabsf(get<1>(a)) == fabsf(get<1>(b)))
                    return get<0>(a) < get<0>(b) ? a : b;
                else
                    return fabsf(get<1>(a)) > fabsf(get<1>(b)) ? a : b;
            };

            auto nid_iterator = make_transform_iterator(counting_iterator<int>(0), placeholders::_1 / n_bins);

            reduce_by_key(
                    cuda::par,
                    nid_iterator, nid_iterator + n_split,
                    make_zip_iterator(make_tuple(counting_iterator<int>(0), gain.device_data())),
                    make_discard_iterator(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max
            );
            LOG(DEBUG) << n_split;
            LOG(DEBUG) << "best rank & gain = " << best_idx_gain;
        }

        //get split points
        {
            const int_float *best_idx_gain_data = best_idx_gain.device_data();
            auto hist_data = hist.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            auto cut_val_data = cut.cut_points_val.device_data();

            sp.resize(n_nodes_in_level);
            auto sp_data = sp.device_data();
            auto nodes_data = tree.nodes.device_data();

            int column_offset = columns.column_offset;

            auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
            device_loop(n_nodes_in_level, [=]__device__(int i) {
                int_float bst = best_idx_gain_data[i];
                float_type best_split_gain = get<1>(bst);
                int split_index = get<0>(bst);
                if (!nodes_data[i + nid_offset].is_valid) {
                    sp_data[i].split_fea_id = -1;
                    sp_data[i].nid = -1;
                    return;
                }
                int fid = hist_fid[split_index];
                sp_data[i].split_fea_id = fid + column_offset;
                sp_data[i].nid = i + nid_offset;
                sp_data[i].gain = fabsf(best_split_gain);
                sp_data[i].fval = cut_val_data[split_index % n_bins];
                sp_data[i].split_bid = (unsigned char) (split_index % n_bins - cut_row_ptr_data[fid]);
                sp_data[i].default_right = best_split_gain < 0;
            });

            for(int i = 0; i < n_nodes_in_level; i++){
                int_float bst = best_idx_gain_data[i];
                float_type best_split_gain = get<1>(bst);
                int split_index = get<0>(bst);
                auto spi_fea_missing_gh_data = sp_data[i].fea_missing_gh.device_data();
                auto spi_rch_sum_gh_data = sp_data[i].rch_sum_gh.device_data();
                device_loop(d_outputs_, [=]__device__(int j){
                    spi_fea_missing_gh_data[j] = missing_gh_data[j * n_partition + i * n_column + hist_fid[split_index]];
                    spi_rch_sum_gh_data[j] = hist_data[j * n_max_splits + split_index];
                }
                );
            }

        }
    }

    LOG(DEBUG) << "split points (gain/fea_id/nid): " << sp;
}

void HistTreeBuilder::update_ins2node_id() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        SyncArray<bool> has_splittable(1);
        auto &columns = shards[device_id].columns;
        //set new node id for each instance
        {
//        TIMED_SCOPE(timerObj, "get new node id");
            auto nid_data = ins2node_id[device_id].device_data();
            const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
            has_splittable.host_data()[0] = false;
            bool *h_s_data = has_splittable.device_data();
            int column_offset = columns.column_offset;

            int n_column = columns.n_column;
            auto dense_bin_id_data = dense_bin_id[device_id].device_data();
            int max_num_bin = param.max_num_bin;
            device_loop(n_instances, [=]__device__(int iid) {
                int nid = nid_data[iid];
                const Tree::TreeNode &node = nodes_data[nid];
                int split_fid = node.split_feature_id;
                if (node.splittable() && ((split_fid - column_offset < n_column) && (split_fid >= column_offset))) {
                    h_s_data[0] = true;
                    unsigned char split_bid = node.split_bid;
                    unsigned char bid = dense_bin_id_data[iid * n_column + split_fid - column_offset];
                    bool to_left = true;
                    if ((bid == max_num_bin && node.default_right) || (bid <= split_bid))
                        to_left = false;
                    if (to_left) {
                        //goes to left child
                        nid_data[iid] = node.lch_index;
                    } else {
                        //right child
                        nid_data[iid] = node.rch_index;
                    }
                }
            });
        }
        LOG(DEBUG) << "new tree_id = " << ins2node_id[device_id];
        has_split[device_id] = has_splittable.host_data()[0];
    });
}

void HistTreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    TreeBuilder::init(dataset, param);
    //TODO refactor
    //init shards
    int n_device = param.n_device;
    shards = vector<Shard>(n_device);
    vector<std::unique_ptr<SparseColumns>> v_columns(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].reset(&shards[i].columns);
        shards[i].ignored_set = SyncArray<bool>(dataset.n_features());
    }
    SparseColumns columns;
    if(dataset.use_cpu)
        columns.csr2csc_cpu(dataset, v_columns);
    else
        columns.csr2csc_gpu(dataset, v_columns);
    cut = vector<HistCut>(param.n_device);
    dense_bin_id = MSyncArray<unsigned char>(param.n_device);
    last_hist = MSyncArray<GHPair>(param.n_device);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        if(dataset.use_cpu)
            cut[device_id].get_cut_points2(shards[device_id].columns, param.max_num_bin, n_instances);
        else
            cut[device_id].get_cut_points3(shards[device_id].columns, param.max_num_bin, n_instances);
        last_hist[device_id].resize((2 << param.depth) * cut[device_id].cut_points_val.size()* dataset.d_outputs_);
    });
    get_bin_ids();
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].release();
    }
    // SyncMem::clear_cache();
    int gpu_num;
    hipError_t err = hipGetDeviceCount(&gpu_num);
    std::atexit([](){
        SyncMem::clear_cache();
    });
}
